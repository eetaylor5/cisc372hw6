#include "hip/hip_runtime.h"
/*
This code is modified for the CISC372 Homework 6 by
Jackson Burns and Emily Taylor.


*/



//Simple optimized box blur
//by: Greg Silber
//Date: 5/1/2021
//This program reads an image and performs a simple averaging of pixels within a supplied radius.  For optimization,
//it does this by computing a running sum for each column within the radius, then averaging that sum.  Then the same for 
//each row.  This should allow it to be easily parallelized by column then by row, since each call is independent.

#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <hip/hip_runtime.h>

//Computes a single row of the destination image by summing radius pixels
//Parameters: src: Teh src image as width*height*bpp 1d array
//            dest: pre-allocated array of size width*height*bpp to receive summed row
//            row: The current row number
//            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
//            rad: the width of the blur
//            bpp: The bits per pixel in the src image
//Returns: None
__global__ void computeRow(float* src,float* dest,int pWidth,int height,int radius,int bpp){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row>=height){
        return;
    }
    int i;
    int bradius=radius*bpp;
    //initialize the first bpp elements so that nothing fails
    for (i=0;i<bpp;i++)
        dest[row*pWidth+i]=src[row*pWidth+i];
    //start the sum up to radius*2 by only adding (nothing to subtract yet)
    for (i=bpp;i<bradius*2*bpp;i++)
        dest[row*pWidth+i]=src[row*pWidth+i]+dest[row*pWidth+i-bpp];
     for (i=bradius*2+bpp;i<pWidth;i++)
        dest[row*pWidth+i]=src[row*pWidth+i]+dest[row*pWidth+i-bpp]-src[row*pWidth+i-2*bradius-bpp];
    //now shift everything over by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    for (i=bradius;i<pWidth;i++){
        dest[row*pWidth+i-bradius]=dest[row*pWidth+i]/(radius*2+1);
    }
    //now the first and last radius values make no sense, so blank them out
    for (i=0;i<bradius;i++){
        dest[row*pWidth+i]=0;
        dest[(row+1)*pWidth-1-i]=0;
    }
}

//Computes a single column of the destination image by summing radius pixels
//Parameters: src: The src image as width*height*bpp 1d array
//            dest: pre-allocated array of size width*height*bpp to receive summed row
//            col: The current column number
//            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
//            height: The height of the source image
//            radius: the width of the blur
//            bpp: The bits per pixel in the src image
//Returns: None
__global__ void computeColumn(uint8_t* src,float* dest,int pWidth,int height,int radius,int bpp){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(col>=pWidth){
        return;
    }
    int i;
    //initialize the first element of each column
    dest[col]=src[col];
    //start tue sum up to radius*2 by only adding
    for (i=1;i<=radius*2;i++)
        dest[i*pWidth+col]=src[i*pWidth+col]+dest[(i-1)*pWidth+col];
    for (i=radius*2+1;i<height;i++)
        dest[i*pWidth+col]=src[i*pWidth+col]+dest[(i-1)*pWidth+col]-src[(i-2*radius-1)*pWidth+col];
    //now shift everything up by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    for (i=radius;i<height;i++){
        dest[(i-radius)*pWidth+col]=dest[i*pWidth+col]/(radius*2+1);
    }
    //now the first and last radius values make no sense, so blank them out
    for (i=0;i<radius;i++){
        dest[i*pWidth+col]=0;
        dest[(height-1)*pWidth-i*pWidth+col]=0;
    }

}

//Usage: Prints the usage for this program
//Parameters: name: The name of the program
//Returns: Always returns -1
int Usage(char* name){
    printf("%s: <filename> <blur radius>\n\tblur radius=pixels to average on any side of the current pixel\n",name);
    return -1;
}

int main(int argc,char** argv){
    long t1,t2;
    int radius=0;
    int i;
    int width,height,bpp,pWidth;
    char* filename;
    uint8_t *img;
    float* dest,*mid;

    if (argc!=3)
        return Usage(argv[0]);
    filename=argv[1];
    sscanf(argv[2],"%d",&radius);
   
    img=stbi_load(filename,&width,&height,&bpp,0);

    pWidth=width*bpp;  //actual width in bytes of an image row

    uint8_t *GPUimg;
    hipMalloc(&GPUimg, sizeof(uint8_t)*pWidth*height);
    hipMemcpy(GPUimg,  img, sizeof(uint8_t)*pWidth*height, hipMemcpyHostToDevice);
    
    hipMalloc(&mid, sizeof(float)*pWidth*height);   
    hipMallocManaged(&dest, sizeof(float)*pWidth*height);
    
    t1=time(NULL);
    computeColumn<<<(pWidth+255)/256, 256>>>(GPUimg,mid,pWidth,height,radius,bpp);
    hipDeviceSynchronize();
    
    computeRow<<<(height+255)/256, 256>>>(mid,dest,pWidth,height,radius,bpp);
    hipDeviceSynchronize();
    
    t2=time(NULL);
    
    //now back to int8 so we can save it
    for (i=0;i<pWidth*height;i++){
        img[i]=(uint8_t)dest[i];
    }
    
    stbi_write_png("output.png",width,height,bpp,img,pWidth);
    stbi_image_free(img);
    hipFree(GPUimg);
    hipFree(dest);
    hipFree(mid);
    
    printf("Blur with radius %d complete in %ld seconds\n",radius,t2-t1);
}
